#define PETSC_SKIP_SPINLOCK
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1

#include <petscconf.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h>   /*I "petscmat.h" I*/
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>
#include <../src/mat/impls/aij/mpi/mpicusparse/mpicusparsematimpl.h>
#include <thrust/advance.h>
#include <petscsf.h>

struct VecCUDAEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<0>(t);
  }
};

static PetscErrorCode MatSetValuesCOO_MPIAIJCUSPARSE(Mat A, const PetscScalar v[], InsertMode imode)
{
  Mat_MPIAIJ         *a = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE *cusp = (Mat_MPIAIJCUSPARSE*)a->spptr;
  PetscInt           n = cusp->coo_nd + cusp->coo_no;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (cusp->coo_p && v) {
    thrust::device_ptr<const PetscScalar> d_v;
    THRUSTARRAY                           *w = NULL;

    if (isCudaMem(v)) {
      d_v = thrust::device_pointer_cast(v);
    } else {
      w = new THRUSTARRAY(n);
      w->assign(v,v+n);
      ierr = PetscLogCpuToGpu(n*sizeof(PetscScalar));CHKERRQ(ierr);
      d_v = w->data();
    }

    auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->coo_p->begin()),
                                                              cusp->coo_pw->begin()));
    auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->coo_p->end()),
                                                              cusp->coo_pw->end()));
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    thrust::for_each(zibit,zieit,VecCUDAEquals());
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
    delete w;
    ierr = MatSetValuesCOO_SeqAIJCUSPARSE(a->A,cusp->coo_pw->data().get(),imode);CHKERRQ(ierr);
    ierr = MatSetValuesCOO_SeqAIJCUSPARSE(a->B,cusp->coo_pw->data().get()+cusp->coo_nd,imode);CHKERRQ(ierr);
  } else {
    ierr = MatSetValuesCOO_SeqAIJCUSPARSE(a->A,v,imode);CHKERRQ(ierr);
    ierr = MatSetValuesCOO_SeqAIJCUSPARSE(a->B,v ? v+cusp->coo_nd : NULL,imode);CHKERRQ(ierr);
  }
  ierr = PetscObjectStateIncrease((PetscObject)A);CHKERRQ(ierr);
  A->num_ass++;
  A->assembled        = PETSC_TRUE;
  A->ass_nonzerostate = A->nonzerostate;
  A->offloadmask      = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}

template <typename Tuple>
struct IsNotOffDiagT
{
  PetscInt _cstart,_cend;

  IsNotOffDiagT(PetscInt cstart, PetscInt cend) : _cstart(cstart), _cend(cend) {}
  __host__ __device__
  inline bool operator()(Tuple t)
  {
    return !(thrust::get<1>(t) < _cstart || thrust::get<1>(t) >= _cend);
  }
};

struct IsOffDiag
{
  PetscInt _cstart,_cend;

  IsOffDiag(PetscInt cstart, PetscInt cend) : _cstart(cstart), _cend(cend) {}
  __host__ __device__
  inline bool operator() (const PetscInt &c)
  {
    return c < _cstart || c >= _cend;
  }
};

struct GlobToLoc
{
  PetscInt _start;

  GlobToLoc(PetscInt start) : _start(start) {}
  __host__ __device__
  inline PetscInt operator() (const PetscInt &c)
  {
    return c - _start;
  }
};

static PetscErrorCode MatSetPreallocationCOO_MPIAIJCUSPARSE(Mat B, PetscInt n, const PetscInt coo_i[], const PetscInt coo_j[])
{
  Mat_MPIAIJ             *b = (Mat_MPIAIJ*)B->data;
  Mat_MPIAIJCUSPARSE     *cusp = (Mat_MPIAIJCUSPARSE*)b->spptr;
  PetscErrorCode         ierr;
  PetscInt               *jj;
  size_t                 noff = 0;
  THRUSTINTARRAY         d_i(n); /* on device, storing partitioned coo_i with diagonal first, and off-diag next */
  THRUSTINTARRAY         d_j(n);
  ISLocalToGlobalMapping l2g;
  hipError_t            cerr;

  PetscFunctionBegin;
  ierr = PetscLayoutSetUp(B->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(B->cmap);CHKERRQ(ierr);
  if (b->A) { ierr = MatCUSPARSEClearHandle(b->A);CHKERRQ(ierr); }
  if (b->B) { ierr = MatCUSPARSEClearHandle(b->B);CHKERRQ(ierr); }
  ierr = PetscFree(b->garray);CHKERRQ(ierr);
  ierr = VecDestroy(&b->lvec);CHKERRQ(ierr);
  ierr = MatDestroy(&b->A);CHKERRQ(ierr);
  ierr = MatDestroy(&b->B);CHKERRQ(ierr);

  ierr = PetscLogCpuToGpu(2.*n*sizeof(PetscInt));CHKERRQ(ierr);
  d_i.assign(coo_i,coo_i+n);
  d_j.assign(coo_j,coo_j+n);
  delete cusp->coo_p;
  delete cusp->coo_pw;
  cusp->coo_p = NULL;
  cusp->coo_pw = NULL;
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  auto firstoffd = thrust::find_if(thrust::device,d_j.begin(),d_j.end(),IsOffDiag(B->cmap->rstart,B->cmap->rend));
  auto firstdiag = thrust::find_if_not(thrust::device,firstoffd,d_j.end(),IsOffDiag(B->cmap->rstart,B->cmap->rend));
  if (firstoffd != d_j.end() && firstdiag != d_j.end()) {
    cusp->coo_p = new THRUSTINTARRAY(n);
    cusp->coo_pw = new THRUSTARRAY(n);
    thrust::sequence(thrust::device,cusp->coo_p->begin(),cusp->coo_p->end(),0);
    auto fzipp = thrust::make_zip_iterator(thrust::make_tuple(d_i.begin(),d_j.begin(),cusp->coo_p->begin()));
    auto ezipp = thrust::make_zip_iterator(thrust::make_tuple(d_i.end(),d_j.end(),cusp->coo_p->end()));
    auto mzipp = thrust::partition(thrust::device,fzipp,ezipp,IsNotOffDiagT<thrust::tuple<PetscInt,PetscInt,PetscInt> >(B->cmap->rstart,B->cmap->rend));
    firstoffd = mzipp.get_iterator_tuple().get<1>();
  }
  cusp->coo_nd = thrust::distance(d_j.begin(),firstoffd);
  cusp->coo_no = thrust::distance(firstoffd,d_j.end());

  /* from global to local */
  thrust::transform(thrust::device,d_i.begin(),d_i.end(),d_i.begin(),GlobToLoc(B->rmap->rstart));
  thrust::transform(thrust::device,d_j.begin(),firstoffd,d_j.begin(),GlobToLoc(B->cmap->rstart));
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

  /* copy offdiag column indices to map on the CPU */
  ierr = PetscMalloc1(cusp->coo_no,&jj);CHKERRQ(ierr); /* jj[] will store compacted col ids of the offdiag part */
  cerr = hipMemcpy(jj,d_j.data().get()+cusp->coo_nd,cusp->coo_no*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  auto o_j = d_j.begin();
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  thrust::advance(o_j,cusp->coo_nd); /* sort and unique offdiag col ids */
  thrust::sort(thrust::device,o_j,d_j.end());
  auto wit = thrust::unique(thrust::device,o_j,d_j.end()); /* return end iter of the unique range */
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  noff = thrust::distance(o_j,wit);
  ierr = PetscMalloc1(noff,&b->garray);CHKERRQ(ierr);
  cerr = hipMemcpy(b->garray,d_j.data().get()+cusp->coo_nd,noff*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  ierr = PetscLogGpuToCpu((noff+cusp->coo_no)*sizeof(PetscInt));CHKERRQ(ierr);
  ierr = ISLocalToGlobalMappingCreate(PETSC_COMM_SELF,1,noff,b->garray,PETSC_COPY_VALUES,&l2g);CHKERRQ(ierr);
  ierr = ISLocalToGlobalMappingSetType(l2g,ISLOCALTOGLOBALMAPPINGHASH);CHKERRQ(ierr);
  ierr = ISGlobalToLocalMappingApply(l2g,IS_GTOLM_DROP,cusp->coo_no,jj,&n,jj);CHKERRQ(ierr);
  if (n != cusp->coo_no) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Unexpected is size %D != %D coo size",n,cusp->coo_no);
  ierr = ISLocalToGlobalMappingDestroy(&l2g);CHKERRQ(ierr);

  ierr = MatCreate(PETSC_COMM_SELF,&b->A);CHKERRQ(ierr);
  ierr = MatSetSizes(b->A,B->rmap->n,B->cmap->n,B->rmap->n,B->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(b->A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->A);CHKERRQ(ierr);
  ierr = MatCreate(PETSC_COMM_SELF,&b->B);CHKERRQ(ierr);
  ierr = MatSetSizes(b->B,B->rmap->n,noff,B->rmap->n,noff);CHKERRQ(ierr);
  ierr = MatSetType(b->B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->B);CHKERRQ(ierr);

  /* GPU memory, cusparse specific call handles it internally */
  ierr = MatSetPreallocationCOO_SeqAIJCUSPARSE(b->A,cusp->coo_nd,d_i.data().get(),d_j.data().get());CHKERRQ(ierr);
  ierr = MatSetPreallocationCOO_SeqAIJCUSPARSE(b->B,cusp->coo_no,d_i.data().get()+cusp->coo_nd,jj);CHKERRQ(ierr);
  ierr = PetscFree(jj);CHKERRQ(ierr);

  ierr = MatCUSPARSESetFormat(b->A,MAT_CUSPARSE_MULT,cusp->diagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetFormat(b->B,MAT_CUSPARSE_MULT,cusp->offdiagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->A,cusp->handle);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->B,cusp->handle);CHKERRQ(ierr);
  /*
  ierr = MatCUSPARSESetStream(b->A,cusp->stream);CHKERRQ(ierr);
  ierr = MatCUSPARSESetStream(b->B,cusp->stream);CHKERRQ(ierr);
  */
  ierr = MatSetUpMultiply_MPIAIJ(B);CHKERRQ(ierr);
  B->preallocated = PETSC_TRUE;
  B->nonzerostate++;

  ierr = MatBindToCPU(b->A,B->boundtocpu);CHKERRQ(ierr);
  ierr = MatBindToCPU(b->B,B->boundtocpu);CHKERRQ(ierr);
  B->offloadmask = PETSC_OFFLOAD_CPU;
  B->assembled = PETSC_FALSE;
  B->was_assembled = PETSC_FALSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE(Mat A,MatReuse scall,IS *glob,Mat *A_loc)
{
  Mat            Ad,Ao;
  const PetscInt *cmap;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMPIAIJGetSeqAIJ(A,&Ad,&Ao,&cmap);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEMergeMats(Ad,Ao,scall,A_loc);CHKERRQ(ierr);
  if (glob) {
    PetscInt cst, i, dn, on, *gidx;

    ierr = MatGetLocalSize(Ad,NULL,&dn);CHKERRQ(ierr);
    ierr = MatGetLocalSize(Ao,NULL,&on);CHKERRQ(ierr);
    ierr = MatGetOwnershipRangeColumn(A,&cst,NULL);CHKERRQ(ierr);
    ierr = PetscMalloc1(dn+on,&gidx);CHKERRQ(ierr);
    for (i=0; i<dn; i++) gidx[i]    = cst + i;
    for (i=0; i<on; i++) gidx[i+dn] = cmap[i];
    ierr = ISCreateGeneral(PetscObjectComm((PetscObject)Ad),dn+on,gidx,PETSC_OWN_POINTER,glob);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatMPIAIJSetPreallocation_MPIAIJCUSPARSE(Mat B,PetscInt d_nz,const PetscInt d_nnz[],PetscInt o_nz,const PetscInt o_nnz[])
{
  Mat_MPIAIJ         *b = (Mat_MPIAIJ*)B->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE*)b->spptr;
  PetscErrorCode     ierr;
  PetscInt           i;

  PetscFunctionBegin;
  ierr = PetscLayoutSetUp(B->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(B->cmap);CHKERRQ(ierr);
  if (PetscDefined(USE_DEBUG) && d_nnz) {
    for (i=0; i<B->rmap->n; i++) {
      if (d_nnz[i] < 0) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"d_nnz cannot be less than 0: local row %D value %D",i,d_nnz[i]);
    }
  }
  if (PetscDefined(USE_DEBUG) && o_nnz) {
    for (i=0; i<B->rmap->n; i++) {
      if (o_nnz[i] < 0) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"o_nnz cannot be less than 0: local row %D value %D",i,o_nnz[i]);
    }
  }
#if defined(PETSC_USE_CTABLE)
  ierr = PetscTableDestroy(&b->colmap);CHKERRQ(ierr);
#else
  ierr = PetscFree(b->colmap);CHKERRQ(ierr);
#endif
  ierr = PetscFree(b->garray);CHKERRQ(ierr);
  ierr = VecDestroy(&b->lvec);CHKERRQ(ierr);
  ierr = VecScatterDestroy(&b->Mvctx);CHKERRQ(ierr);
  /* Because the B will have been resized we simply destroy it and create a new one each time */
  ierr = MatDestroy(&b->B);CHKERRQ(ierr);
  if (!b->A) {
    ierr = MatCreate(PETSC_COMM_SELF,&b->A);CHKERRQ(ierr);
    ierr = MatSetSizes(b->A,B->rmap->n,B->cmap->n,B->rmap->n,B->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->A);CHKERRQ(ierr);
  }
  if (!b->B) {
    PetscMPIInt size;
    ierr = MPI_Comm_size(PetscObjectComm((PetscObject)B),&size);CHKERRMPI(ierr);
    ierr = MatCreate(PETSC_COMM_SELF,&b->B);CHKERRQ(ierr);
    ierr = MatSetSizes(b->B,B->rmap->n,size > 1 ? B->cmap->N : 0,B->rmap->n,size > 1 ? B->cmap->N : 0);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)B,(PetscObject)b->B);CHKERRQ(ierr);
  }
  ierr = MatSetType(b->A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSetType(b->B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatBindToCPU(b->A,B->boundtocpu);CHKERRQ(ierr);
  ierr = MatBindToCPU(b->B,B->boundtocpu);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation(b->A,d_nz,d_nnz);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation(b->B,o_nz,o_nnz);CHKERRQ(ierr);
  ierr = MatCUSPARSESetFormat(b->A,MAT_CUSPARSE_MULT,cusparseStruct->diagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetFormat(b->B,MAT_CUSPARSE_MULT,cusparseStruct->offdiagGPUMatFormat);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->A,cusparseStruct->handle);CHKERRQ(ierr);
  ierr = MatCUSPARSESetHandle(b->B,cusparseStruct->handle);CHKERRQ(ierr);
  /* Let A, B use b's handle with pre-set stream
  ierr = MatCUSPARSESetStream(b->A,cusparseStruct->stream);CHKERRQ(ierr);
  ierr = MatCUSPARSESetStream(b->B,cusparseStruct->stream);CHKERRQ(ierr);
  */
  B->preallocated = PETSC_TRUE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatMult_MPIAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_MPIAIJ     *a = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;
  PetscInt       nt;

  PetscFunctionBegin;
  ierr = VecGetLocalSize(xx,&nt);CHKERRQ(ierr);
  if (nt != A->cmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Incompatible partition of A (%D) and xx (%D)",A->cmap->n,nt);
  ierr = VecScatterBegin(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->A->ops->mult)(a->A,xx,yy);CHKERRQ(ierr);
  ierr = VecScatterEnd(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->B->ops->multadd)(a->B,a->lvec,yy,yy);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatZeroEntries_MPIAIJCUSPARSE(Mat A)
{
  Mat_MPIAIJ     *l = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatZeroEntries(l->A);CHKERRQ(ierr);
  ierr = MatZeroEntries(l->B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultAdd_MPIAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_MPIAIJ     *a = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;
  PetscInt       nt;

  PetscFunctionBegin;
  ierr = VecGetLocalSize(xx,&nt);CHKERRQ(ierr);
  if (nt != A->cmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Incompatible partition of A (%D) and xx (%D)",A->cmap->n,nt);
  ierr = VecScatterBegin(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->A->ops->multadd)(a->A,xx,yy,zz);CHKERRQ(ierr);
  ierr = VecScatterEnd(a->Mvctx,xx,a->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  ierr = (*a->B->ops->multadd)(a->B,a->lvec,zz,zz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTranspose_MPIAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_MPIAIJ     *a = (Mat_MPIAIJ*)A->data;
  PetscErrorCode ierr;
  PetscInt       nt;

  PetscFunctionBegin;
  ierr = VecGetLocalSize(xx,&nt);CHKERRQ(ierr);
  if (nt != A->rmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Incompatible partition of A (%D) and xx (%D)",A->rmap->n,nt);
  ierr = (*a->B->ops->multtranspose)(a->B,xx,a->lvec);CHKERRQ(ierr);
  ierr = (*a->A->ops->multtranspose)(a->A,xx,yy);CHKERRQ(ierr);
  ierr = VecScatterBegin(a->Mvctx,a->lvec,yy,ADD_VALUES,SCATTER_REVERSE);CHKERRQ(ierr);
  ierr = VecScatterEnd(a->Mvctx,a->lvec,yy,ADD_VALUES,SCATTER_REVERSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSESetFormat_MPIAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_MPIAIJ         *a               = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE * cusparseStruct = (Mat_MPIAIJCUSPARSE*)a->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT_DIAG:
    cusparseStruct->diagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_MULT_OFFDIAG:
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseStruct->diagGPUMatFormat    = format;
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. Only MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_DIAG, and MAT_CUSPARSE_MULT_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSetFromOptions_MPIAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  MatCUSPARSEStorageFormat format;
  PetscErrorCode           ierr;
  PetscBool                flg;
  Mat_MPIAIJ               *a = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE       *cusparseStruct = (Mat_MPIAIJCUSPARSE*)a->spptr;

  PetscFunctionBegin;
  ierr = PetscOptionsHead(PetscOptionsObject,"MPIAIJCUSPARSE options");CHKERRQ(ierr);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_diag_storage_format","sets storage format of the diagonal blocks of (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->diagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT_DIAG,format);CHKERRQ(ierr);
    }
    ierr = PetscOptionsEnum("-mat_cusparse_mult_offdiag_storage_format","sets storage format of the off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->offdiagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT_OFFDIAG,format);CHKERRQ(ierr);
    }
    ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of the diagonal and off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparseStruct->diagGPUMatFormat,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
    }
  }
  ierr = PetscOptionsTail();CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatAssemblyEnd_MPIAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode     ierr;
  Mat_MPIAIJ         *mpiaij = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE *cusp = (Mat_MPIAIJCUSPARSE*)mpiaij->spptr;
  PetscObjectState   onnz = A->nonzerostate;

  PetscFunctionBegin;
  ierr = MatAssemblyEnd_MPIAIJ(A,mode);CHKERRQ(ierr);
  if (mpiaij->lvec) { ierr = VecSetType(mpiaij->lvec,VECSEQCUDA);CHKERRQ(ierr); }
  if (onnz != A->nonzerostate && cusp->deviceMat) {
    PetscSplitCSRDataStructure d_mat = cusp->deviceMat, h_mat;
    hipError_t                cerr;

    ierr = PetscInfo(A,"Destroy device mat since nonzerostate changed\n");CHKERRQ(ierr);
    ierr = PetscNew(&h_mat);CHKERRQ(ierr);
    cerr = hipMemcpy(h_mat,d_mat,sizeof(*d_mat),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    cerr = hipFree(h_mat->colmap);CHKERRCUDA(cerr);
    cerr = hipFree(d_mat);CHKERRCUDA(cerr);
    ierr = PetscFree(h_mat);CHKERRQ(ierr);
    cusp->deviceMat = NULL;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatDestroy_MPIAIJCUSPARSE(Mat A)
{
  PetscErrorCode     ierr;
  Mat_MPIAIJ         *aij            = (Mat_MPIAIJ*)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE*)aij->spptr;
  hipsparseStatus_t   stat;

  PetscFunctionBegin;
  if (!cusparseStruct) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_COR,"Missing spptr");
  if (cusparseStruct->deviceMat) {
    PetscSplitCSRDataStructure d_mat = cusparseStruct->deviceMat, h_mat;
    hipError_t                cerr;

    ierr = PetscInfo(A,"Have device matrix\n");CHKERRQ(ierr);
    ierr = PetscNew(&h_mat);CHKERRQ(ierr);
    cerr = hipMemcpy(h_mat,d_mat,sizeof(*d_mat),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    cerr = hipFree(h_mat->colmap);CHKERRCUDA(cerr);
    cerr = hipFree(d_mat);CHKERRCUDA(cerr);
    ierr = PetscFree(h_mat);CHKERRQ(ierr);
  }
  try {
    if (aij->A) { ierr = MatCUSPARSEClearHandle(aij->A);CHKERRQ(ierr); }
    if (aij->B) { ierr = MatCUSPARSEClearHandle(aij->B);CHKERRQ(ierr); }
    stat = hipsparseDestroy(cusparseStruct->handle);CHKERRCUSPARSE(stat);
    /* We want cusparseStruct to use PetscDefaultCudaStream
    if (cusparseStruct->stream) {
      hipError_t err = hipStreamDestroy(cusparseStruct->stream);CHKERRCUDA(err);
    }
    */
    delete cusparseStruct->coo_p;
    delete cusparseStruct->coo_pw;
    delete cusparseStruct;
  } catch(char *ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"Mat_MPIAIJCUSPARSE error: %s", ex);
  }
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatMPIAIJSetPreallocation_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatMPIAIJGetLocalMatMerge_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetFormat_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatConvert_mpiaijcusparse_hypre_C",NULL);CHKERRQ(ierr);
  ierr = MatDestroy_MPIAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatConvert_MPIAIJ_MPIAIJCUSPARSE(Mat B, MatType mtype, MatReuse reuse, Mat* newmat)
{
  PetscErrorCode     ierr;
  Mat_MPIAIJ         *a;
  Mat_MPIAIJCUSPARSE *cusparseStruct;
  hipsparseStatus_t   stat;
  Mat                A;

  PetscFunctionBegin;
  if (reuse == MAT_INITIAL_MATRIX) {
    ierr = MatDuplicate(B,MAT_COPY_VALUES,newmat);CHKERRQ(ierr);
  } else if (reuse == MAT_REUSE_MATRIX) {
    ierr = MatCopy(B,*newmat,SAME_NONZERO_PATTERN);CHKERRQ(ierr);
  }
  A = *newmat;
  A->boundtocpu = PETSC_FALSE;
  ierr = PetscFree(A->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&A->defaultvectype);CHKERRQ(ierr);

  a = (Mat_MPIAIJ*)A->data;
  if (a->A) { ierr = MatSetType(a->A,MATSEQAIJCUSPARSE);CHKERRQ(ierr); }
  if (a->B) { ierr = MatSetType(a->B,MATSEQAIJCUSPARSE);CHKERRQ(ierr); }
  if (a->lvec) {
    ierr = VecSetType(a->lvec,VECSEQCUDA);CHKERRQ(ierr);
  }

  if (reuse != MAT_REUSE_MATRIX && !a->spptr) {
    a->spptr = new Mat_MPIAIJCUSPARSE;

    cusparseStruct                      = (Mat_MPIAIJCUSPARSE*)a->spptr;
    cusparseStruct->diagGPUMatFormat    = MAT_CUSPARSE_CSR;
    cusparseStruct->offdiagGPUMatFormat = MAT_CUSPARSE_CSR;
    cusparseStruct->coo_p               = NULL;
    cusparseStruct->coo_pw              = NULL;
    cusparseStruct->stream              = 0;
    cusparseStruct->deviceMat           = NULL;
    stat = hipsparseCreate(&(cusparseStruct->handle));CHKERRCUSPARSE(stat);
  }

  A->ops->assemblyend           = MatAssemblyEnd_MPIAIJCUSPARSE;
  A->ops->mult                  = MatMult_MPIAIJCUSPARSE;
  A->ops->multadd               = MatMultAdd_MPIAIJCUSPARSE;
  A->ops->multtranspose         = MatMultTranspose_MPIAIJCUSPARSE;
  A->ops->setfromoptions        = MatSetFromOptions_MPIAIJCUSPARSE;
  A->ops->destroy               = MatDestroy_MPIAIJCUSPARSE;
  A->ops->zeroentries           = MatZeroEntries_MPIAIJCUSPARSE;
  A->ops->productsetfromoptions = MatProductSetFromOptions_MPIAIJBACKEND;

  ierr = PetscObjectChangeTypeName((PetscObject)A,MATMPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatMPIAIJGetLocalMatMerge_C",MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatMPIAIJSetPreallocation_C",MatMPIAIJSetPreallocation_MPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetFormat_C",MatCUSPARSESetFormat_MPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",MatSetPreallocationCOO_MPIAIJCUSPARSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",MatSetValuesCOO_MPIAIJCUSPARSE);CHKERRQ(ierr);
#if defined(PETSC_HAVE_HYPRE)
  ierr = PetscObjectComposeFunction((PetscObject)A,"MatConvert_mpiaijcusparse_hypre_C",MatConvert_AIJ_HYPRE);CHKERRQ(ierr);
#endif
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatCreate_MPIAIJCUSPARSE(Mat A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscCUDAInitializeCheck();CHKERRQ(ierr);
  ierr = MatCreate_MPIAIJ(A);CHKERRQ(ierr);
  ierr = MatConvert_MPIAIJ_MPIAIJCUSPARSE(A,MATMPIAIJCUSPARSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*@
   MatCreateAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format).  This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradigm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATMPIAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt M,PetscInt N,PetscInt d_nz,const PetscInt d_nnz[],PetscInt o_nz,const PetscInt o_nnz[],Mat *A)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,M,N);CHKERRQ(ierr);
  ierr = MPI_Comm_size(comm,&size);CHKERRMPI(ierr);
  if (size > 1) {
    ierr = MatSetType(*A,MATMPIAIJCUSPARSE);CHKERRQ(ierr);
    ierr = MatMPIAIJSetPreallocation(*A,d_nz,d_nnz,o_nz,o_nnz);CHKERRQ(ierr);
  } else {
    ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    ierr = MatSeqAIJSetPreallocation(*A,d_nz,d_nnz);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   MATAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as MATMPIAIJCUSPARSE.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   This matrix type is identical to MATSEQAIJCUSPARSE when constructed with a single process communicator,
   and MATMPIAIJCUSPARSE otherwise.  As a result, for single process communicators,
   MatSeqAIJSetPreallocation is supported, and similarly MatMPIAIJSetPreallocation is supported
   for communicators controlling multiple processes.  It is recommended that you call both of
   the above preallocation routines for simplicity.

   Options Database Keys:
+  -mat_type mpiaijcusparse - sets the matrix type to "mpiaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of diagonal and off-diagonal matrices during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_diag_storage_format csr - sets the storage format of diagonal matrix during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_offdiag_storage_format csr - sets the storage format of off-diagonal matrix during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

 .seealso: MatCreateAIJCUSPARSE(), MATSEQAIJCUSPARSE, MATMPIAIJCUSPARSE, MatCreateSeqAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

/*MC
   MATMPIAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as MATAIJCUSPARSE.

  Level: beginner

 .seealso: MATAIJCUSPARSE, MATSEQAIJCUSPARSE
M*/

PETSC_INTERN PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat);

// get GPU pointers to stripped down Mat. For both seq and MPI Mat.
PetscErrorCode MatCUSPARSEGetDeviceMatWrite(Mat A, PetscSplitCSRDataStructure *B)
{
  PetscSplitCSRDataStructure d_mat;
  PetscMPIInt                size;
  PetscErrorCode             ierr;
  int                        *ai = NULL,*bi = NULL,*aj = NULL,*bj = NULL;
  PetscScalar                *aa = NULL,*ba = NULL;
  Mat_SeqAIJ                 *jaca = NULL;
  Mat_SeqAIJCUSPARSE         *cusparsestructA = NULL;
  CsrMatrix                  *matrixA = NULL,*matrixB = NULL;

  PetscFunctionBegin;
  if (!A->assembled) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Need already assembled matrix");
  if (A->factortype != MAT_FACTOR_NONE) {
    *B = NULL;
    PetscFunctionReturn(0);
  }
  ierr = MPI_Comm_size(PetscObjectComm((PetscObject)A),&size);CHKERRMPI(ierr);
  if (size == 1) {
    PetscBool isseqaij;

    ierr = PetscObjectBaseTypeCompare((PetscObject)A,MATSEQAIJ,&isseqaij);CHKERRQ(ierr);
    if (isseqaij) {
      jaca = (Mat_SeqAIJ*)A->data;
      if (!jaca->roworiented) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Device assembly does not currently support column oriented values insertion");
      cusparsestructA = (Mat_SeqAIJCUSPARSE*)A->spptr;
      d_mat = cusparsestructA->deviceMat;
      ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
    } else {
      Mat_MPIAIJ *aij = (Mat_MPIAIJ*)A->data;
      if (!aij->roworiented) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Device assembly does not currently support column oriented values insertion");
      Mat_MPIAIJCUSPARSE *spptr = (Mat_MPIAIJCUSPARSE*)aij->spptr;
      jaca = (Mat_SeqAIJ*)aij->A->data;
      cusparsestructA = (Mat_SeqAIJCUSPARSE*)aij->A->spptr;
      d_mat = spptr->deviceMat;
      ierr = MatSeqAIJCUSPARSECopyToGPU(aij->A);CHKERRQ(ierr);
    }
    if (cusparsestructA->format==MAT_CUSPARSE_CSR) {
      Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestructA->mat;
      if (!matstruct) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing Mat_SeqAIJCUSPARSEMultStruct for A");
      matrixA = (CsrMatrix*)matstruct->mat;
      bi = NULL;
      bj = NULL;
      ba = NULL;
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Device Mat needs MAT_CUSPARSE_CSR");
  } else {
    Mat_MPIAIJ *aij = (Mat_MPIAIJ*)A->data;
    if (!aij->roworiented) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Device assembly does not currently support column oriented values insertion");
    jaca = (Mat_SeqAIJ*)aij->A->data;
    Mat_SeqAIJ *jacb = (Mat_SeqAIJ*)aij->B->data;
    Mat_MPIAIJCUSPARSE *spptr = (Mat_MPIAIJCUSPARSE*)aij->spptr;

    if (!A->nooffprocentries && !aij->donotstash) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Device assembly does not currently support offproc values insertion. Use MatSetOption(A,MAT_NO_OFF_PROC_ENTRIES,PETSC_TRUE) or MatSetOption(A,MAT_IGNORE_OFF_PROC_ENTRIES,PETSC_TRUE)");
    cusparsestructA = (Mat_SeqAIJCUSPARSE*)aij->A->spptr;
    Mat_SeqAIJCUSPARSE *cusparsestructB = (Mat_SeqAIJCUSPARSE*)aij->B->spptr;
    if (cusparsestructA->format!=MAT_CUSPARSE_CSR) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Device Mat A needs MAT_CUSPARSE_CSR");
    if (cusparsestructB->format==MAT_CUSPARSE_CSR) {
      ierr = MatSeqAIJCUSPARSECopyToGPU(aij->A);CHKERRQ(ierr);
      ierr = MatSeqAIJCUSPARSECopyToGPU(aij->B);CHKERRQ(ierr);
      Mat_SeqAIJCUSPARSEMultStruct *matstructA = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestructA->mat;
      Mat_SeqAIJCUSPARSEMultStruct *matstructB = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestructB->mat;
      if (!matstructA) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing Mat_SeqAIJCUSPARSEMultStruct for A");
      if (!matstructB) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing Mat_SeqAIJCUSPARSEMultStruct for B");
      matrixA = (CsrMatrix*)matstructA->mat;
      matrixB = (CsrMatrix*)matstructB->mat;
      if (jacb->compressedrow.use) {
        if (!cusparsestructB->rowoffsets_gpu) {
          cusparsestructB->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n+1);
          cusparsestructB->rowoffsets_gpu->assign(jacb->i,jacb->i+A->rmap->n+1);
        }
        bi = thrust::raw_pointer_cast(cusparsestructB->rowoffsets_gpu->data());
      } else {
        bi = thrust::raw_pointer_cast(matrixB->row_offsets->data());
      }
      bj = thrust::raw_pointer_cast(matrixB->column_indices->data());
      ba = thrust::raw_pointer_cast(matrixB->values->data());
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Device Mat B needs MAT_CUSPARSE_CSR");
    d_mat = spptr->deviceMat;
  }
  if (jaca->compressedrow.use) {
    if (!cusparsestructA->rowoffsets_gpu) {
      cusparsestructA->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n+1);
      cusparsestructA->rowoffsets_gpu->assign(jaca->i,jaca->i+A->rmap->n+1);
    }
    ai = thrust::raw_pointer_cast(cusparsestructA->rowoffsets_gpu->data());
  } else {
    ai = thrust::raw_pointer_cast(matrixA->row_offsets->data());
  }
  aj = thrust::raw_pointer_cast(matrixA->column_indices->data());
  aa = thrust::raw_pointer_cast(matrixA->values->data());

  if (!d_mat) {
    hipError_t                cerr;
    PetscSplitCSRDataStructure h_mat;

    // create and populate strucy on host and copy on device
    ierr = PetscInfo(A,"Create device matrix\n");CHKERRQ(ierr);
    ierr = PetscNew(&h_mat);CHKERRQ(ierr);
    cerr = hipMalloc((void**)&d_mat,sizeof(*d_mat));CHKERRCUDA(cerr);
    if (size > 1) { /* need the colmap array */
      Mat_MPIAIJ *aij = (Mat_MPIAIJ*)A->data;
      int        *colmap;
      PetscInt   ii,n = aij->B->cmap->n,N = A->cmap->N;

      if (n && !aij->garray) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"MPIAIJ Matrix was assembled but is missing garray");

      ierr = PetscCalloc1(N+1,&colmap);CHKERRQ(ierr);
      for (ii=0; ii<n; ii++) colmap[aij->garray[ii]] = (int)(ii+1);

      h_mat->offdiag.i = bi;
      h_mat->offdiag.j = bj;
      h_mat->offdiag.a = ba;
      h_mat->offdiag.n = A->rmap->n;

      cerr = hipMalloc((void**)&h_mat->colmap,(N+1)*sizeof(int));CHKERRCUDA(cerr);
      cerr = hipMemcpy(h_mat->colmap,colmap,(N+1)*sizeof(int),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
      ierr = PetscFree(colmap);CHKERRQ(ierr);
    }
    h_mat->rstart = A->rmap->rstart;
    h_mat->rend   = A->rmap->rend;
    h_mat->cstart = A->cmap->rstart;
    h_mat->cend   = A->cmap->rend;
    h_mat->N      = A->cmap->N;
    h_mat->diag.i = ai;
    h_mat->diag.j = aj;
    h_mat->diag.a = aa;
    h_mat->diag.n = A->rmap->n;
    h_mat->rank   = PetscGlobalRank;
    // copy pointers and metadata to device
    cerr = hipMemcpy(d_mat,h_mat,sizeof(*d_mat),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscFree(h_mat);CHKERRQ(ierr);
  } else {
    ierr = PetscInfo(A,"Reusing device matrix\n");CHKERRQ(ierr);
  }
  *B = d_mat;
  A->offloadmask = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}
