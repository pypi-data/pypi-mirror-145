#include "hip/hip_runtime.h"
#include "query_knn.cuh"

using namespace lbvh;

__global__ void query_knn_kernel(const BVHNode *nodes,
                                 const float3* __restrict__ points,
                                 const unsigned int* __restrict__ sorted_indices,
                                 const unsigned int root_index,
                                 const float max_radius,
                                 const float3* __restrict__ queries,
                                 const unsigned int N,
                                 // custom parameters
                                 unsigned int* indices_out,
                                 float* distances_out,
                                 unsigned int* n_neighbors_out
                                 )
{
    unsigned int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (query_idx >= N)
        return;
    StaticPriorityQueue<float, K> queue(max_radius);
    query_knn(nodes, points, sorted_indices, root_index, &queries[query_idx], queue);
    __syncwarp(); // synchronize the warp before the write operation
    queue.write_results(&indices_out[query_idx * K], &distances_out[query_idx * K], &n_neighbors_out[query_idx]); // write back the results
}